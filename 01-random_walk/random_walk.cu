#include "hip/hip_runtime.h"
// Copyright © 2014 Martin Ueding <dev@martin-ueding.de>

#include "random_walk.h"

#include <math.h>

__global__
void init_kernel(int walker_count, int *walkers, float *distances_dev, hiprandState_t * curand_states) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= walker_count) {
        return;
    }

    walkers[idx] = 0.0;
    walkers[idx + 1] = 0.0;

    hiprand_init(0, idx, 0, curand_states + idx);
}

__global__
void random_walk_kernel(int walker_count, int steps, int *walkers, float *distances_dev, hiprandState_t * curand_states) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= walker_count) {
        return;
    }

    hiprandState_t state = curand_states[idx];

    // Copy variables into a local register to avoid costly global memory
    // accesses.
    int x = walkers[idx];
    int y = walkers[idx + 1];

    for (int step = 0; step != steps; step++) {
        int random = hiprand(&state) % 4;

        if (random < 0) {
            random = -random;
        }

        // XXX This is probably a bad implementation since a lot of branching
        // will slow it down.
        if (random == 0) {
            x++;
        }
        else if (random == 1) {
            x--;
        }
        else if (random == 2) {
            y++;
        }
        else if (random == 3) {
            y--;
        }
    }

    walkers[idx] = x;
    walkers[idx + 1] = y;
    curand_states[idx] = state;

    float square_distance = x*x + y*y;

    distances_dev[idx] = sqrt(square_distance);
}
